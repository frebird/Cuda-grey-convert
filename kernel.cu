#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<iostream>

#include <iostream>  
#include <opencv2/core/core.hpp>  
#include <opencv2/highgui/highgui.hpp> 

#define CHANNELS 3

__global__ void colorToGreyScaleConversion(unsigned char *pout, unsigned char *pin, int width, int height) {
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int Row = blockIdx.y*blockDim.y + threadIdx.y;

	if (Col < width && Row < height) {
		int greyoffset = Row * width + Col;
		int rgbOffset = greyoffset * CHANNELS;

		unsigned char r = pin[rgbOffset];
		unsigned char g = pin[rgbOffset + 1];
		unsigned char b = pin[rgbOffset + 2];

		pout[greyoffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	}
}

using namespace cv;
int main(void) {

	// 读入一张图片（缩小图）    
	Mat img = imread("E:\\opencv\\lena512color.tiff");
	// 创建一个名为 "图片"窗口    
	namedWindow("lena");
	// 在窗口中显示图片   
	imshow("lena", img);
	// 等待6000 ms后窗口自动关闭    
	waitKey(6000);

	const int imgheight = img.rows;
	const int imgwidth = img.cols;
	const int imgchannel = img.channels();

	Mat grayImage(imgheight, imgwidth, CV_8UC1, Scalar(0));

	unsigned char *dev_pin;
	unsigned char *dev_pout;

	hipMalloc((void**)&dev_pin, imgheight*imgwidth*imgchannel* sizeof(unsigned char));
	hipMalloc((void**)&dev_pout, imgheight*imgwidth*sizeof(unsigned char));

	hipMemcpy(dev_pin, img.data, imgheight*imgwidth*imgchannel * sizeof(unsigned char), hipMemcpyHostToDevice);


	dim3 BlockDim(16, 16);
	dim3 GridDim((imgwidth - 1) / BlockDim.x + 1, (imgheight - 1) / BlockDim.y + 1);
	colorToGreyScaleConversion << <GridDim, BlockDim >> > (dev_pout, dev_pin, imgwidth, imgheight);

	hipMemcpy(grayImage.data, dev_pout, imgheight*imgwidth*sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(dev_pin);
	hipFree(dev_pout);
	imshow("grayImage", grayImage);
	waitKey(3000);
	return 0;
}






